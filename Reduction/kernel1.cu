#include<hip/hip_runtime.h>



#define BLOCK_SIZE 256

// avoid warp divergence
// n/2->0, n/4->0, n/8->0
__global__ void __kernel_reduce_sum(float *A, int N, float *res){

    int start_id = blockIdx.x*BLOCK_SIZE+ threadIdx.x;
    int tid = threadIdx.x;
    //add the grid 
    float val = 0;
    for(int i=start_id; i<N; i+=gridDim.x*BLOCK_SIZE){
        val += A[i];
    }
    __shared__ float data[BLOCK_SIZE];
    
    data[tid] = val;
    

    __syncthreads();

    for(int offset=BLOCK_SIZE/2; offset>0; offset>>=1){
        if(tid<offset){
            data[tid]+=data[tid+offset];
        }
        __syncthreads();
    }
    if(tid==0)
        res[blockIdx.x] = data[0];

  



}





void custom_reduce_sum(float *A, int N, float *res, int block_num){
    dim3 block(BLOCK_SIZE);
    dim3 grid(block_num); //m*BLOCK_SIZE<=N, m<=BLOCK_SIZE

    __kernel_reduce_sum<<<grid, block>>>(A, N, res);
    // cudaDeviceSynchronize();
    // __kernel_reduce_sum<<<1, block>>>(res, block_num, res);

}